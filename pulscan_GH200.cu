#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <omp.h>

extern "C" {
#include "localcdflib.h"
}

double log_asymtotic_incomplete_gamma(double a, double z)
/*
  log_asymtotic_incomplete_gamma(double a, double z):
      Return the natural log of the incomplete gamma function in
          its asymtotic limit as z->infty.  This is from Abramowitz
          and Stegun eqn 6.5.32.
*/
{
    double x = 1.0, newxpart = 1.0, term = 1.0;
    int ii = 1;

    //printf("log_asymtotic_incomplete_gamma() being called with arguments:\n");
    //printf("   a = %f, z = %f\n", a, z);

    while (fabs(newxpart) > 1e-15) {
        term *= (a - ii);
        newxpart = term / pow(z, ii);
        x += newxpart;
        ii += 1;
        //printf("ii = %d, x = %f, newxpart = %f\n", ii, x, newxpart);
    }
    //printf("Took %d iterations.\n", ii);
    return (a - 1.0) * log(z) - z + log(x);
}

double log_asymtotic_gamma(double z)
/*
  log_asymtotic_gamma(double z):
      Return the natural log of the gamma function in its asymtotic limit
          as z->infty.  This is from Abramowitz and Stegun eqn 6.1.41.
*/
{
    double x, y;
    //printf("log_asymtotic_gamma() being called with argument z = %f\n", z);
    x = (z - 0.5) * log(z) - z + 0.91893853320467267;
    y = 1.0 / (z * z);
    x += (((-5.9523809523809529e-4 * y
            + 7.9365079365079365079365e-4) * y
           - 2.7777777777777777777778e-3) * y + 8.3333333333333333333333e-2) / z;
    return x;
}
double extended_equiv_gaussian_sigma(double logp)
/*
  extended_equiv_gaussian_sigma(double logp):
      Return the equivalent gaussian sigma corresponding to the 
          natural log of the cumulative gaussian probability logp.
          In other words, return x, such that Q(x) = p, where Q(x)
          is the cumulative normal distribution.  This version uses
          the rational approximation from Abramowitz and Stegun,
          eqn 26.2.23.  Using the log(P) as input gives a much
          extended range.
*/
{
    double t, num, denom;

    t = sqrt(-2.0 * logp);
    num = 2.515517 + t * (0.802853 + t * 0.010328);
    denom = 1.0 + t * (1.432788 + t * (0.189269 + t * 0.001308));
    return t - num / denom;
}


double chi2_logp(double chi2, double dof)
/* MODIFIED FOR PULSCAN TO CLOSE INVALID REGION AT HIGH DOF */
/* Return the natural log probability corresponding to a chi^2 value */
/* of chi2 given dof degrees of freedom. */
{
    double logp;
    //printf("chi2 = %f, dof = %f\n", chi2, dof);

    if (chi2 <= 0.0) {
        return -INFINITY;
    }
    //printf("chi2/dof = %f\n", chi2/dof);
    // COMMENT OUT NEXT LINE IS THE MODIFICATION
    //if (chi2 / dof > 15.0 || (dof > 150 && chi2 / dof > 6.0)) {
    if (chi2 / dof > 1.0) {
        //printf("chi2/dof > 1.0\n");
        // printf("Using asymtotic expansion...\n");
        // Use some asymtotic expansions for the chi^2 distribution
        //   this is eqn 26.4.19 of A & S
        logp = log_asymtotic_incomplete_gamma(0.5 * dof, 0.5 * chi2) -
            log_asymtotic_gamma(0.5 * dof);
    } else {
        //printf("chi2/dof <= 1.0\n");
        int which, status;
        double p, q, bound, df = dof, x = chi2;

        which = 1;
        status = 0;
        // Determine the basic probability
        cdfchi(&which, &p, &q, &x, &df, &status, &bound);
        if (status) {
            printf("\nError in cdfchi() (chi2_logp()):\n");
            printf("   status = %d, bound = %g\n", status, bound);
            printf("   p = %g, q = %g, x = %g, df = %g\n\n", p, q, x, df);
            exit(1);
        }
        // printf("p = %.3g  q = %.3g\n", p, q);
        logp = log(q);
    }
    return logp;
}

double equivalent_gaussian_sigma(double logp)
/* Return the approximate significance in Gaussian sigmas */
/* corresponding to a natural log probability logp        */
{
    double x;

    if (logp < -600.0) {
        x = extended_equiv_gaussian_sigma(logp);
    } else {
        int which, status;
        double p, q, bound, mean = 0.0, sd = 1.0;
        q = exp(logp);
        p = 1.0 - q;
        which = 2;
        status = 0;
        /* Convert to a sigma */
        cdfnor(&which, &p, &q, &x, &mean, &sd, &status, &bound);
        if (status) {
            if (status == -2) {
                x = 0.0;
            } else if (status == -3) {
                x = 38.5;
            } else {
                printf("\nError in cdfnor() (candidate_sigma()):\n");
                printf("   status = %d, bound = %g\n", status, bound);
                printf("   p = %g, q = %g, x = %g, mean = %g, sd = %g\n\n",
                       p, q, x, mean, sd);
                exit(1);
            }
        }
    }
    if (x < 0.0)
        return 0.0;
    else
        return x;
}

double candidate_sigma(double power, int numsum, double numtrials)
/* Return the approximate significance in Gaussian       */
/* sigmas of a candidate of numsum summed powers,        */
/* taking into account the number of independent trials. */
{
    //printf("candidate_sigma() being called with arguments:\n");
    //printf("   power = %f, numsum = %d, numtrials = %f\n",
    //       power, numsum, numtrials);
    double logp, chi2, dof;

    if (power <= 0.0) {
        return 0.0;
    }

    // Get the natural log probability
    chi2 = 2.0 * power;
    dof = 2.0 * numsum;
    logp = chi2_logp(chi2, dof);

    // Correct for numtrials
    logp += log(numtrials);

    // Convert to sigma
    return equivalent_gaussian_sigma(logp);
}

/*
 *  This Quickselect routine is based on the algorithm described in
 *  "Numerical recipies in C", Second Edition,
 *  Cambridge University Press, 1992, Section 8.5, ISBN 0-521-43108-5
*/

/* Fast computation of the median of an array. */
/* Note:  It messes up the order!              */

#define ELEM_SWAP(a,b) { register float t=(a);(a)=(b);(b)=t; }

float median_function(float arr[], int n)
{
    int low, high;
    int median;
    int middle, ll, hh;

    low = 0;
    high = n - 1;
    median = (low + high) / 2;
    for (;;) {
        if (high <= low)        /* One element only */
            return arr[median];

        if (high == low + 1) {  /* Two elements only */
            if (arr[low] > arr[high])
                ELEM_SWAP(arr[low], arr[high]);
            return arr[median];
        }

        /* Find median of low, middle and high items; swap into position low */
        middle = (low + high) / 2;
        if (arr[middle] > arr[high])
            ELEM_SWAP(arr[middle], arr[high]);
        if (arr[low] > arr[high])
            ELEM_SWAP(arr[low], arr[high]);
        if (arr[middle] > arr[low])
            ELEM_SWAP(arr[middle], arr[low]);

        /* Swap low item (now in position middle) into position (low+1) */
        ELEM_SWAP(arr[middle], arr[low + 1]);

        /* Nibble from each end towards middle, swapping items when stuck */
        ll = low + 1;
        hh = high;
        for (;;) {
            do
                ll++;
            while (arr[low] > arr[ll]);
            do
                hh--;
            while (arr[hh] > arr[low]);

            if (hh < ll)
                break;

            ELEM_SWAP(arr[ll], arr[hh]);
        }

        /* Swap middle item (in position low) back into correct position */
        ELEM_SWAP(arr[low], arr[hh]);

        /* Re-set active partition */
        if (hh <= median)
            low = ll;
        if (hh >= median)
            high = hh - 1;
    }
}

#undef ELEM_SWAP

void normalize_block_quickselect(float* block, size_t block_size) {
    if (block_size == 0) return;

    // Allocate memory for a copy of the block
    float* sorted_block = (float*) malloc(sizeof(float) * block_size);

    // Copy the block to sorted_block
    memcpy(sorted_block, block, sizeof(float) * block_size);

    // Compute the median using the new function
    float median = median_function(sorted_block, block_size);
    //printf("Median: %f\n", median);

    // Compute the MAD
    for (size_t i = 0; i < block_size; i++) {
        sorted_block[i] = fabs(sorted_block[i] - median); // Calculate the absolute deviation from the median
    }

    // Re-compute the median of the deviations to get the MAD
    float mad = median_function(sorted_block, block_size);
    //printf("MAD: %f\n", mad);

    // Free the allocated memory
    free(sorted_block);

    // Scale the MAD by the constant scale factor k
    float k = 1.4826f; // Scale factor to convert MAD to standard deviation for a normal distribution
    mad *= k;

    // Normalize the block
    if (mad != 0) {
        for (size_t i = 0; i < block_size; i++) {
            block[i] = (block[i] - median) / mad;
        }
    }
}

// TODO CUDA streams




struct candidate{
    float power;
    float logp;
    int r;
    int z;
    int numharm;
};

double __device__ power_to_logp(float chi2, float dof){
    double double_dof = (double) dof;
    double double_chi2 = (double) chi2;
    // Use boundary condition
    if (dof >= chi2 * 1.05){
        return 0.0;
    } else {
        double x = 1500 * double_dof / double_chi2;
        // Updated polynomial equation
        double f_x = (-4.460405902717228e-46 * pow(x, 16) + 9.492786384945832e-42 * pow(x, 15) - 
               9.147045144529116e-38 * pow(x, 14) + 5.281085384219971e-34 * pow(x, 13) - 
               2.0376166670276118e-30 * pow(x, 12) + 5.548033164083744e-27 * pow(x, 11) - 
               1.0973877021703706e-23 * pow(x, 10) + 1.5991806841151474e-20 * pow(x, 9) - 
               1.7231488066853853e-17 * pow(x, 8) + 1.3660070957914896e-14 * pow(x, 7) - 
               7.861795249869729e-12 * pow(x, 6) + 3.2136336591718867e-09 * pow(x, 5) - 
               9.046641813341226e-07 * pow(x, 4) + 0.00016945948004599545 * pow(x, 3) - 
               0.0214942314851717 * pow(x, 2) + 2.951595476316614 * x - 
               755.240918031251);
        double logp = chi2 * f_x / 1500;
        return logp;
    }
}

// takes a 1D array like this:
// magnitudeSquaredArray:   [0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0]
// and adds these elements together, effectively performing a harmonic sum
// decimatedArray2:         [0,0,0,0,0,x,0,0,0,0,0,x,x,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0]
// decimatedArray3:         [0,0,0,0,0,x,0,0,0,0,0,x,x,0,0,0,0,x,x,x,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0]
// decimatedArray4:         [0,0,0,0,0,x,0,0,0,0,0,x,x,0,0,0,0,x,x,x,0,0,0,x,x,x,x,0,0,0,0,0,0,0,0,0]
//                                     |<--------->|<--------->|<--------->|
//                                        equal spacing between harmonics

__global__ void decimateHarmonics(float* magnitudeSquaredArray, float* decimatedArray2, float* decimatedArray3, float* decimatedArray4, long numMagnitudes){
    int globalThreadIndex = blockDim.x*blockIdx.x + threadIdx.x;

    float fundamental;
    float harmonic1a, harmonic1b;
    float harmonic2a, harmonic2b, harmonic2c;
    float harmonic3a, harmonic3b, harmonic3c, harmonic3d;

    if (globalThreadIndex*2+1 < numMagnitudes){
        fundamental = magnitudeSquaredArray[globalThreadIndex];
        harmonic1a = magnitudeSquaredArray[globalThreadIndex*2];
        harmonic1b = magnitudeSquaredArray[globalThreadIndex*2+1];
        decimatedArray2[globalThreadIndex] = fundamental+harmonic1a+harmonic1b;
    }

    if (globalThreadIndex*3+2 < numMagnitudes){
        harmonic2a = magnitudeSquaredArray[globalThreadIndex*3];
        harmonic2b = magnitudeSquaredArray[globalThreadIndex*3+1];
        harmonic2c = magnitudeSquaredArray[globalThreadIndex*3+2];
        decimatedArray3[globalThreadIndex] = fundamental+harmonic1a+harmonic1b
                                                +harmonic2a+harmonic2b+harmonic2c;
    }

    if (globalThreadIndex*4+3 < numMagnitudes){
        harmonic3a = magnitudeSquaredArray[globalThreadIndex*4];
        harmonic3b = magnitudeSquaredArray[globalThreadIndex*4+1];
        harmonic3c = magnitudeSquaredArray[globalThreadIndex*4+2];
        harmonic3d = magnitudeSquaredArray[globalThreadIndex*4+3];
        decimatedArray4[globalThreadIndex] = fundamental+harmonic1a+harmonic1b
                                                +harmonic2a+harmonic2b+harmonic2c
                                                +harmonic3a+harmonic3b+harmonic3c+harmonic3d;
    }

    //if (globalThreadIndex == 50000){
    //    printf("fundamental: %f, harmonic1a: %f, harmonic1b: %f, harmonic2a: %f, harmonic2b: %f, harmonic2c: %f, harmonic3a: %f, harmonic3b: %f, harmonic3c: %f, harmonic3d: %f\n", fundamental, harmonic1a, harmonic1b, harmonic2a, harmonic2b, harmonic2c, harmonic3a, harmonic3b, harmonic3c, harmonic3d);
    //}
}

// logarithmic zstep, zmax = 256, numThreads = 256
// TODO: any zmax
// TODO: any blockWidth
// will need to use dynamically allocated shared memory
__global__ void boxcarFilterArray(float* magnitudeSquaredArray, candidate* globalCandidateArray, int numharm, long numFloats){
    __shared__ float lookupArray[512];
    __shared__ float sumArray[256];
    __shared__ float searchArray[256];
    __shared__ candidate localCandidateArray[16];

    int globalThreadIndex = blockDim.x*blockIdx.x + threadIdx.x;
    int localThreadIndex = threadIdx.x;

    lookupArray[localThreadIndex] = magnitudeSquaredArray[globalThreadIndex];
    lookupArray[localThreadIndex + 256] = magnitudeSquaredArray[globalThreadIndex + 256];

    __syncthreads();

    // initialise the sum array
    sumArray[localThreadIndex] = lookupArray[localThreadIndex];
    __syncthreads();
    // begin boxcar filtering
    int targetZ = 1;
    int outputCounter = 0;

    for (int z = 0; z < 256; z+=1){
        sumArray[localThreadIndex] +=  lookupArray[localThreadIndex + z];
        if (z = targetZ){
            searchArray[localThreadIndex] = sumArray[localThreadIndex];
            for (int stride = blockDim.x / 2; stride>0; stride >>= 1){
                if (localThreadIndex < stride){
                    searchArray[localThreadIndex] = fmaxf(searchArray[localThreadIndex], searchArray[localThreadIndex + stride]);
                }
                __syncthreads();
            }
            localCandidateArray[outputCounter].power = searchArray[0];
            localCandidateArray[outputCounter].r = blockIdx.x*blockDim.x;
            localCandidateArray[outputCounter].z = z;
            localCandidateArray[outputCounter].logp = 0.0f;
            localCandidateArray[outputCounter].numharm = numharm;
            outputCounter+=1;
            targetZ *= 2;
        }
        __syncthreads();
    }

    __syncthreads();

    if (localThreadIndex < 16){
        globalCandidateArray[blockIdx.x*16+localThreadIndex] = localCandidateArray[localThreadIndex];
    }
}

__global__ void calculateLogp(candidate* globalCandidateArray, long numCandidates, int numSum){
    int globalThreadIndex = blockDim.x*blockIdx.x + threadIdx.x;
    if (globalThreadIndex < numCandidates){
        double logp = power_to_logp(globalCandidateArray[globalThreadIndex].power,globalCandidateArray[globalThreadIndex].z*numSum);
        globalCandidateArray[globalThreadIndex].logp = (float) logp;
    }
}

void copyDeviceArrayToHostAndPrint(float* deviceArray, long numFloats){
    float* hostArray;
    hostArray = (float*)malloc(sizeof(float)*numFloats);
    hipMemcpy(hostArray, deviceArray, sizeof(float)*numFloats,hipMemcpyDeviceToHost);
    for (int i = 0; i < numFloats; i++){
        printf("%f\n", hostArray[i]);
    }
    free(hostArray);
}

void copyDeviceArrayToHostAndSaveToFile(float* deviceArray, long numFloats, const char* filename){
    float* hostArray;
    hostArray = (float*)malloc(sizeof(float)*numFloats);
    hipMemcpy(hostArray, deviceArray, sizeof(float)*numFloats,hipMemcpyDeviceToHost);
    FILE *f = fopen(filename, "wb");
    // write in csv format, one number per column
    for (int i = 0; i < numFloats; i++){
        fprintf(f, "%f\n", hostArray[i]);
    }
    fclose(f);
    free(hostArray);
}

float* compute_magnitude_block_normalization_mad(const char *filepath, int *magnitude_size, int ncpus) {
    // begin timer for reading input file
    double start = omp_get_wtime();
    size_t block_size = 32768; // needs to be much larger than max boxcar width

    //printf("Reading file: %s\n", filepath);

    FILE *f = fopen(filepath, "rb");
    if (f == NULL) {
        perror("Error opening file");
        return NULL;
    }

    // Determine the size of the file
    fseek(f, 0, SEEK_END);
    long filesize = ftell(f);
    fseek(f, 0, SEEK_SET);

    size_t num_floats = filesize / sizeof(float);

    // Allocate memory for the data
    float* data = (float*) malloc(sizeof(float) * num_floats);
    if(data == NULL) {
        printf("Memory allocation failed\n");
        fclose(f);
        return NULL;
    }
    
    size_t n = fread(data, sizeof(float), num_floats, f);
    if (n % 2 != 0) {
        printf("Data file does not contain an even number of floats\n");
        fclose(f);
        free(data);
        return NULL;
    }

    size_t size = n / 2;
    float* magnitude = (float*) malloc(sizeof(float) * size);
    if(magnitude == NULL) {
        printf("Memory allocation failed\n");
        free(data);
        return NULL;
    }

    double end = omp_get_wtime();
    double time_spent = end - start;
    printf("Reading the data took      %f seconds using 1 thread\n", time_spent);

    start = omp_get_wtime();

    #pragma omp parallel for
    // Perform block normalization
    for (size_t block_start = 0; block_start < size; block_start += block_size) {
        size_t block_end = block_start + block_size < size ? block_start + block_size : size;
        size_t current_block_size = block_end - block_start;

        // Separate the real and imaginary parts
        float* real_block = (float*) malloc(sizeof(float) * current_block_size);
        float* imag_block = (float*) malloc(sizeof(float) * current_block_size);

        if (real_block == NULL || imag_block == NULL) {
            printf("Memory allocation failed for real_block or imag_block\n");
            free(real_block);
            free(imag_block);
        }

        for (size_t i = 0; i < current_block_size; i++) {
            real_block[i] = data[2 * (block_start + i)];
            imag_block[i] = data[2 * (block_start + i) + 1];
        }

        // Normalize real and imaginary parts independently
        normalize_block_quickselect(real_block, current_block_size);
        normalize_block_quickselect(imag_block, current_block_size);

        // Recompute the magnitudes after normalization
        for (size_t i = block_start; i < block_end; i++) {
            magnitude[i] = real_block[i - block_start] * real_block[i - block_start] +
                        imag_block[i - block_start] * imag_block[i - block_start];
        }

        free(real_block);
        free(imag_block);
    }

    magnitude[0] = 0.0f; // set DC component of magnitude spectrum to 0

    fclose(f);
    free(data);

    *magnitude_size = (int) size;

    end = omp_get_wtime();
    time_spent = end - start;
    printf("Normalizing the data took  %f seconds using %d thread(s)\n", time_spent, ncpus);
    return magnitude;
}


#define RESET   "\033[0m"
#define FLASHING   "\033[5m"
#define BOLD   "\033[1m"

const char* frame = 
"      .     *    +    .      .   .                 .              *   +  \n"
"  *     " BOLD "____        __" RESET "  +   .             .       " BOLD "__________  __  __" RESET "    .\n"
".   +  " BOLD "/ __ \\__  __/ /_____________" RESET "*" BOLD "_____" RESET "     .  " BOLD "/ ____/ __ \\/ / / /" RESET "     \n"
"      " BOLD "/ /_/ / / / / / ___/ ___/ __ `/ __ \\______/ / __/ /_/ / / / /" RESET "  +   .\n"
"   . " BOLD "/ ____/ /_/ / (__  ) /__/ /_/ / / / /_____/ /_/ / ____/ /_/ /" RESET "\n"
"    " BOLD "/_/" RESET "  . " BOLD "\\__,_/_/____/\\___/\\__,_/_/ /_/" RESET "   *  " BOLD "\\____/_/    \\____/ .  " FLASHING "*" RESET "\n"
" .    .       .   +           .        .         +        .       .      .\n"
"  +     .        .      +       .           .            .    +    .\n"
"        J. White, K. Adámek, J. Roy, S. Ransom, W. Armour   2023\n\n";

int main(int argc, char* argv[]){
    int debug = 0;
    printf("%s", frame);

    // start high resolution timer to measure gpu initialisation time using chrono
    auto start_chrono = std::chrono::high_resolution_clock::now();
    
    hipDeviceSynchronize();

    auto end_chrono = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_chrono - start_chrono);
    
    printf("GPU initialisation took:                %f ms\n",(float)duration.count());
    
    // start timing
    start_chrono = std::chrono::high_resolution_clock::now();

    if (argc < 2) {
        printf("Please provide the input file path as a command line argument.\n");
        return 1;
    }

    int ncpus = 72;

    // define filepath variable
    const char* filepath = argv[1];

    int magnitude_array_size;
    float* magnitudes = compute_magnitude_block_normalization_mad(filepath, &magnitude_array_size, ncpus);
    if (magnitudes == NULL) {
        printf("Error reading the input file.\n");
        return 1;
    }

    // stop timing
    end_chrono = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_chrono - start_chrono);
    printf("Magnitude took:                         %f ms\n",(float)duration.count());


    int numMagnitudes = magnitude_array_size;

    float* magnitudeSquaredArray;
    hipMalloc((void**)&magnitudeSquaredArray, sizeof(float)*numMagnitudes);

    // start timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);


    hipMemcpy(magnitudeSquaredArray, magnitudes, sizeof(float)*numMagnitudes, hipMemcpyHostToDevice);



    //copyDeviceArrayToHostAndPrint(magnitudeSquaredArray, numMagnitudes);
    //copyDeviceArrayToHostAndSaveToFile(magnitudeSquaredArray, numMagnitudes, "magnitudeSquaredArray.bin");

    // stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Magnitude took:                         %f ms\n", milliseconds);

    // start timing
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    float* decimatedArrayBy2;
    float* decimatedArrayBy3;
    float* decimatedArrayBy4;
    hipMalloc((void**)&decimatedArrayBy2, sizeof(float)*numMagnitudes/2);
    hipMalloc((void**)&decimatedArrayBy3, sizeof(float)*numMagnitudes/3);
    hipMalloc((void**)&decimatedArrayBy4, sizeof(float)*numMagnitudes/4);

    int numThreadsDecimate = 256;
    int numBlocksDecimate = (numMagnitudes/2 + numThreadsDecimate - 1)/ numThreadsDecimate;


    if (debug == 1) {
        printf("Calling decimateHarmonics with %d blocks and %d threads per block\n", numBlocksDecimate, numThreadsDecimate);
    }
    decimateHarmonics<<<numBlocksDecimate, numThreadsDecimate>>>(magnitudeSquaredArray, decimatedArrayBy2, decimatedArrayBy3, decimatedArrayBy4, numMagnitudes);
    hipDeviceSynchronize();
    
    // stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Decimation took:                        %f ms\n", milliseconds);

    // start timing
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int numThreadsBoxcar = 256;
    int numBlocksBoxcar1 = (numMagnitudes + numThreadsBoxcar - 1)/ numThreadsBoxcar;
    int numBlocksBoxcar2 = (numMagnitudes/2 + numThreadsBoxcar - 1)/ numThreadsBoxcar;
    int numBlocksBoxcar3 = (numMagnitudes/3 + numThreadsBoxcar - 1)/ numThreadsBoxcar;
    int numBlocksBoxcar4 = (numMagnitudes/4 + numThreadsBoxcar - 1)/ numThreadsBoxcar;

    candidate* globalCandidateArray1;
    candidate* globalCandidateArray2;
    candidate* globalCandidateArray3;
    candidate* globalCandidateArray4;

    hipMalloc((void**)&globalCandidateArray1, sizeof(candidate)*16*numBlocksBoxcar1);
    hipMalloc((void**)&globalCandidateArray2, sizeof(candidate)*16*numBlocksBoxcar2);
    hipMalloc((void**)&globalCandidateArray3, sizeof(candidate)*16*numBlocksBoxcar3);
    hipMalloc((void**)&globalCandidateArray4, sizeof(candidate)*16*numBlocksBoxcar4);

    
    if (debug == 1) {
        printf("Calling boxcarFilterArray with %d blocks and %d threads per block\n", numBlocksBoxcar1, numThreadsBoxcar);
        printf("Calling boxcarFilterArray with %d blocks and %d threads per block\n", numBlocksBoxcar2, numThreadsBoxcar);
        printf("Calling boxcarFilterArray with %d blocks and %d threads per block\n", numBlocksBoxcar3, numThreadsBoxcar);
        printf("Calling boxcarFilterArray with %d blocks and %d threads per block\n", numBlocksBoxcar4, numThreadsBoxcar);
    }
    boxcarFilterArray<<<numBlocksBoxcar1, numThreadsBoxcar>>>(magnitudeSquaredArray, globalCandidateArray1, 1, numMagnitudes);
    boxcarFilterArray<<<numBlocksBoxcar2, numThreadsBoxcar>>>(decimatedArrayBy2, globalCandidateArray2, 2, numMagnitudes/2);
    boxcarFilterArray<<<numBlocksBoxcar3, numThreadsBoxcar>>>(decimatedArrayBy3, globalCandidateArray3, 3, numMagnitudes/3);
    boxcarFilterArray<<<numBlocksBoxcar4, numThreadsBoxcar>>>(decimatedArrayBy4, globalCandidateArray4, 4, numMagnitudes/4);
    hipDeviceSynchronize();

    // stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Boxcar filtering took:                  %f ms\n", milliseconds);

    // start timing
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int numThreadsLogp = 256;
    int numBlocksLogp1 = (numBlocksBoxcar1*16 + numThreadsLogp - 1)/ numThreadsLogp;
    int numBlocksLogp2 = (numBlocksBoxcar2*16 + numThreadsLogp - 1)/ numThreadsLogp;
    int numBlocksLogp3 = (numBlocksBoxcar3*16 + numThreadsLogp - 1)/ numThreadsLogp;
    int numBlocksLogp4 = (numBlocksBoxcar4*16 + numThreadsLogp - 1)/ numThreadsLogp;

    if (debug == 1) {
        printf("Calling calculateLogp with %d blocks and %d threads per block\n", numBlocksLogp1, numThreadsLogp);
        printf("Calling calculateLogp with %d blocks and %d threads per block\n", numBlocksLogp2, numThreadsLogp);
        printf("Calling calculateLogp with %d blocks and %d threads per block\n", numBlocksLogp3, numThreadsLogp);
        printf("Calling calculateLogp with %d blocks and %d threads per block\n", numBlocksLogp4, numThreadsLogp);
    }
    calculateLogp<<<numBlocksLogp1, numThreadsLogp>>>(globalCandidateArray1, numBlocksBoxcar1*16, 1);
    calculateLogp<<<numBlocksLogp2, numThreadsLogp>>>(globalCandidateArray2, numBlocksBoxcar2*16, 3);
    calculateLogp<<<numBlocksLogp3, numThreadsLogp>>>(globalCandidateArray3, numBlocksBoxcar3*16, 6);
    calculateLogp<<<numBlocksLogp4, numThreadsLogp>>>(globalCandidateArray4, numBlocksBoxcar4*16, 10);
    hipDeviceSynchronize();

    // stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Logp time taken:                        %f ms\n", milliseconds);

    // start chrono timer for writing output file
    start_chrono = std::chrono::high_resolution_clock::now();

    candidate* hostCandidateArray1;
    candidate* hostCandidateArray2;
    candidate* hostCandidateArray3;
    candidate* hostCandidateArray4;

    hostCandidateArray1 = (candidate*)malloc(sizeof(candidate)*16*numBlocksBoxcar1);
    hostCandidateArray2 = (candidate*)malloc(sizeof(candidate)*16*numBlocksBoxcar2);
    hostCandidateArray3 = (candidate*)malloc(sizeof(candidate)*16*numBlocksBoxcar3);
    hostCandidateArray4 = (candidate*)malloc(sizeof(candidate)*16*numBlocksBoxcar4);

    hipMemcpy(hostCandidateArray1, globalCandidateArray1, sizeof(candidate)*16*numBlocksBoxcar1, hipMemcpyDeviceToHost);
    hipMemcpy(hostCandidateArray2, globalCandidateArray2, sizeof(candidate)*16*numBlocksBoxcar2, hipMemcpyDeviceToHost);
    hipMemcpy(hostCandidateArray3, globalCandidateArray3, sizeof(candidate)*16*numBlocksBoxcar3, hipMemcpyDeviceToHost);
    hipMemcpy(hostCandidateArray4, globalCandidateArray4, sizeof(candidate)*16*numBlocksBoxcar4, hipMemcpyDeviceToHost);

    // output filename is inputfilename with the .fft stripped and replaced with .gpucand
    char outputFilename[256];
    strncpy(outputFilename, filepath, strlen(filepath) - 4);
    outputFilename[strlen(filepath) - 4] = '\0';
    strcat(outputFilename, ".gpucand");


    // write the candidates to a csv file with a header line
    //FILE *csvFile = fopen("gpucandidates.csv", "w");
    FILE *csvFile = fopen(outputFilename, "w");
    fprintf(csvFile, "r,z,power,logp,numharm\n");

    float logpThreshold = -50;

    for (int i = 0; i < numBlocksBoxcar1*16; i++){
        if (i % 16 < 9){
            if (hostCandidateArray1[i].logp < logpThreshold){
                if (hostCandidateArray1[i].r != 0){
                    fprintf(csvFile, "%d,%d,%f,%f,%d\n", hostCandidateArray1[i].r, hostCandidateArray1[i].z, hostCandidateArray1[i].power, hostCandidateArray1[i].logp, hostCandidateArray1[i].numharm);
                }
            }
        }
    }
    
    for (int i = 0; i < numBlocksBoxcar2*16; i++){
        if (i % 16 < 9){
            if (hostCandidateArray2[i].logp < logpThreshold){
                if (hostCandidateArray1[i].r != 0){
                    fprintf(csvFile, "%d,%d,%f,%f,%d\n", hostCandidateArray2[i].r, hostCandidateArray2[i].z, hostCandidateArray2[i].power, hostCandidateArray2[i].logp, hostCandidateArray2[i].numharm);
                }
            }
        }
    }

    for (int i = 0; i < numBlocksBoxcar3*16; i++){
        if (i % 16 < 9){
            if (hostCandidateArray3[i].logp < logpThreshold){
                if (hostCandidateArray1[i].r != 0){
                    fprintf(csvFile, "%d,%d,%f,%f,%d\n", hostCandidateArray3[i].r, hostCandidateArray3[i].z, hostCandidateArray3[i].power, hostCandidateArray3[i].logp, hostCandidateArray3[i].numharm);
                }
            }
        }
    }

    for (int i = 0; i < numBlocksBoxcar4*16; i++){
        if (i % 16 < 9){
            if (hostCandidateArray4[i].logp < logpThreshold){
                if (hostCandidateArray1[i].r != 0){
                    fprintf(csvFile, "%d,%d,%f,%f,%d\n", hostCandidateArray4[i].r, hostCandidateArray4[i].z, hostCandidateArray4[i].power, hostCandidateArray4[i].logp, hostCandidateArray4[i].numharm);
                }
            }
        }
    }

    fclose(csvFile);

    // stop chrono timer for writing output file
    end_chrono = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_chrono - start_chrono);
    printf("Writing output file took:               %f ms\n", (float)duration.count());

    hipFree(magnitudeSquaredArray);
    hipFree(decimatedArrayBy2);
    hipFree(decimatedArrayBy3);
    hipFree(decimatedArrayBy4);
    hipFree(globalCandidateArray1);
    hipFree(globalCandidateArray2);
    hipFree(globalCandidateArray3);
    hipFree(globalCandidateArray4);

    // check last cuda error
    hipError_t error = hipGetLastError();
    if (error != hipSuccess){
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }

    return 0;
}

